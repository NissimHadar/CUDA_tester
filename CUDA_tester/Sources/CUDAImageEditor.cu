#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "CUDAImageEditor.cuh"


__global__
void removeBlue(const unsigned int width, const unsigned char* const inputPixels, unsigned char* const outputPixels) {
	// Set third byte to 0
	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	unsigned int byteIndex = 3 * (x + width * y);

	// Use weighted values
	int colour = (0.2125 * inputPixels[byteIndex + 0]) + (0.7154 * inputPixels[byteIndex + 1]) + (0.0721 * inputPixels[byteIndex + 2]);

	outputPixels[byteIndex + 0] = colour;
	outputPixels[byteIndex + 1] = colour;
	outputPixels[byteIndex + 2] = colour;
}

void CUDAImageEditor::convertToMonochrome(const unsigned int height, const unsigned int width, const unsigned char* const h_inputPixels, unsigned char* const h_outputPixels) {
	const unsigned int BUFFER_SIZE{ height * width * 3 };

	// Put pixel buffer in device memory
	unsigned char* d_inputPixels;
	unsigned char* d_outputPixels;
	hipMalloc(&d_inputPixels, BUFFER_SIZE);
	hipMalloc(&d_outputPixels, BUFFER_SIZE);

	hipMemcpy(d_inputPixels, h_inputPixels, BUFFER_SIZE, hipMemcpyHostToDevice);

	// Blocks will be 8x8 threads
	dim3 threadsPerBlock(8, 8);

	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);

	removeBlue<<< numBlocks, threadsPerBlock >>>(width, d_inputPixels, d_outputPixels);

	hipMemcpy((void *)h_outputPixels, d_outputPixels, BUFFER_SIZE, hipMemcpyDeviceToHost);

	hipFree(d_inputPixels);
	hipFree(d_outputPixels);
}